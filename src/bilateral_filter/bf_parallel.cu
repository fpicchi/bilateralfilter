#include "hip/hip_runtime.h"
/**
 * @file bf_parallel.cu
 * @author Federico Picchi
 * @brief CUDA-based implementation of Bilateral Filter
 * @date June 2021
 * 
 * @copyright Copyright (c) 2021
 * 
 */


#include "bilateral_filter/bf.hpp"
#if defined(_DEBUG)
#include <iostream>
#endif

inline void handleError(hipError_t err, int line) {
#if defined(_DEBUG)
    if (err) {
        std::cerr << __FILE__ << ": ERROR " << err << " CALLING CUDA FUNCTION IN LINE: " << line << "\n";
        exit(1);
    }
#endif
    return;
}

__global__
void bf_parallel_k(const uchar*  const source, 
                       uchar*  const destination, 
                 const int           diameter,
                 const double* const gi,
                 const double* const gs,
                 const int*    const space_coord,
                 const int           maxk,
                 const int           width,
                 const int           height,
                 const size_t        s_step,
                 const size_t        d_step) {
    // Shared memory setup
    extern __shared__ double shared[];
    double* const gi_s = (double*)shared;
    double* const gs_s = gi_s + 256;
    int*    const space_coord_s = (int*)&gs_s[diameter * diameter];
    uchar*  const tile_s = (uchar*)&space_coord_s[diameter * diameter];
    // Ids and vals setup
    const int radius = diameter / 2;
    const int global_j = (int)(threadIdx.x + blockIdx.x * blockDim.x) - radius * (int)(1 + 2 * blockIdx.x);
    const int global_i = (int)(threadIdx.y + blockIdx.y * blockDim.y) - radius * (int)(1 + 2 * blockIdx.y);
    const int sharedId = threadIdx.y * blockDim.x + threadIdx.x;
    // Copy from global memory to shared memory
    if (sharedId < 256)
        gi_s[sharedId] = gi[sharedId];
    if (sharedId < diameter * diameter) {
        space_coord_s[sharedId] = space_coord[sharedId];
        gs_s[sharedId] = gs[sharedId];
    }
    if (global_i >= height + radius || global_j >= width + radius) return;
    tile_s[sharedId] = source[(global_i + radius) * s_step + radius + global_j];
    if (global_i >= height || global_j >= width) return;
    if (threadIdx.x < radius || threadIdx.x >= blockDim.x - radius || 
        threadIdx.y < radius || threadIdx.y >= blockDim.y - radius)
        return;
    __syncthreads();

    // Calc new pixel value
    double sum = 0, wsum = 0;
    const int val0 = tile_s[sharedId]; //< Center of the template.
    for (int k = 0; k < maxk; k++)
    {
        const int val = tile_s[sharedId + space_coord_s[k]];
        // The weight is gaussian space * color space.
        const double w = gs_s[k] * gi_s[abs(val - val0)];
        sum += val * w;
        wsum += w;
    }
    destination[global_j + global_i * d_step] = (uchar)lround(sum / wsum);
}

inline size_t calcBytesNeeded(const int blockSize, const int diameter) {
    return (size_t)blockSize + 256 * sizeof(double) + diameter * diameter * (sizeof(int) + sizeof(double));
}

cv::Mat bf_parallel(const cv::Mat &source, 
    const int diameter, const double sigma_i, const double sigma_s)
{
    const int radius = diameter / 2;
    //Calculate optimal CUDA configuration
    int blockSize = 1024;
    int minGridSize;
    size_t bytesNeeded = calcBytesNeeded(blockSize, diameter);
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       bf_parallel_k, bytesNeeded, 0);
    // Recalc SM bytes needed
    bytesNeeded = calcBytesNeeded(blockSize, diameter);
    // Round up according to matrix size
    blockSize = (int)sqrt(blockSize);
    dim3 blockSize_2D(blockSize, blockSize);
    blockSize -= 2*radius;
    dim3 gridSize_2D((source.cols + blockSize - 1) / blockSize, (source.rows + blockSize - 1) / blockSize);
    // Create destination matrix
    cv::Mat dst = cv::Mat::zeros(source.rows, source.cols, CV_8U);
    // Create an image with a border.
    cv::Mat temp;
    cv::copyMakeBorder(source, temp, radius, radius, radius, radius,
        cv::BorderTypes::BORDER_REFLECT_101);
    // Init color weight.
    double coeff_i = -0.5 / (sigma_i * sigma_i);
    std::vector<double> gi_vec(256);
    double *gi = &gi_vec[0];
    for (int i = 0; i < 256; i++)
        gi[i] = exp(i * i * coeff_i);
    // Generate gaussian space.
    std::vector<double> gs_vec(diameter * diameter);
    std::vector<int> space_coord_vec(diameter * diameter); //< Save here coord.
    double *gs = &gs_vec[0];
    int    *space_coord = &space_coord_vec[0];
    const double coeff_s = -0.5 / (sigma_s * sigma_s);
    int maxk = 0;
    for (int i = -radius; i <= radius; i++)
    {
        for (int j = -radius; j <= radius; j++)
        {
            double r = sqrt(i * i + j * j);
            if (r > radius) //< Circle.
                continue;
            gs[maxk] = exp(r * r * coeff_s);
            space_coord[maxk++] = i * (int)blockSize_2D.x + j;
        }
    }
    // Copy data to device
    uchar* temp_d;
    handleError(hipMalloc(&temp_d, temp.total()), __LINE__);
    handleError(hipMemcpy(temp_d, temp.data, temp.total(), hipMemcpyHostToDevice), __LINE__);
    uchar* dst_d;
    handleError(hipMalloc(&dst_d, dst.total()), __LINE__);
    handleError(hipMemcpy(dst_d, dst.data, dst.total(), hipMemcpyHostToDevice), __LINE__);
    double* gs_d;
    handleError(hipMalloc(&gs_d, diameter * diameter * sizeof(double)), __LINE__);
    handleError(hipMemcpy(gs_d, gs, diameter * diameter * sizeof(double), hipMemcpyHostToDevice), __LINE__);
    double* gi_d;
    handleError(hipMalloc(&gi_d, 256 * sizeof(double)), __LINE__);
    handleError(hipMemcpy(gi_d, gi, 256 * sizeof(double), hipMemcpyHostToDevice), __LINE__);
    int* space_coord_d;
    handleError(hipMalloc(&space_coord_d, diameter * diameter * sizeof(int)), __LINE__);
    handleError(hipMemcpy(space_coord_d, space_coord, diameter * diameter * sizeof(int), hipMemcpyHostToDevice), __LINE__);
    
    // Filtering process
    bf_parallel_k << < gridSize_2D, blockSize_2D, bytesNeeded >> > (temp_d, dst_d, diameter, gi_d, gs_d,
                                                space_coord_d, maxk, (int)source.cols,
                                                (int)source.rows, temp.step, dst.step);

    handleError(hipDeviceSynchronize(), __LINE__);
    
    // Copy data from device
    handleError(hipMemcpy(dst.data, dst_d, dst.total(), hipMemcpyDeviceToHost), __LINE__);
    handleError(hipFree(temp_d), __LINE__);
    handleError(hipFree(dst_d), __LINE__);
    handleError(hipFree(gs_d), __LINE__);
    handleError(hipFree(gi_d), __LINE__);
    handleError(hipFree(space_coord_d), __LINE__);

    return dst;
}