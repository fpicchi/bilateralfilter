#include "hip/hip_runtime.h"
/**
 * @file bf_parallel.cu
 * @author Federico Picchi
 * @brief CUDA-based implementation of Bilateral Filter
 * @date June 2021
 * 
 * @copyright Copyright (c) 2021
 * 
 */


#include "bilateral_filter/bf.hpp"
#if defined(_DEBUG)
#include <iostream>
#endif

inline void handleError(hipError_t err, int line) {
#if defined(_DEBUG)
    if (err) {
        std::cerr << __FILE__ << ": ERROR " << err << " CALLING CUDA FUNCTION IN LINE: " << line << "\n";
        exit(1);
    }
#endif
    return;
}

__global__
void bf_parallel_naive_k(const uchar*  const source, 
                       uchar*  const destination, 
                 const int           radius,
                 const double* const gi,
                 const double* const gs,
                 const int*    const space_coord,
                 const int           maxk,
                 const size_t        width,
                 const size_t        height,
                 const size_t        s_step,
                 const size_t        d_step) {
    const int global_j = threadIdx.x + blockIdx.x * blockDim.x;
    const int global_i = threadIdx.y + blockIdx.y * blockDim.y;
    if (global_i >= height || global_j >= width) return;
    const uchar* const sptr = source + (global_i + radius) * s_step + radius;
    uchar* const dptr = destination + global_i * d_step;

    double sum = 0, wsum = 0;
    const int val0 = sptr[global_j]; //< Center of the template.
    for (int k = 0; k < maxk; k++)
    {
        const int val = sptr[global_j + space_coord[k]];
        // The weight is gaussian space * color space.
        const double w = gs[k] * gi[abs(val - val0)];
        sum += val * w;
        wsum += w;
    }
    dptr[global_j] = (uchar)lround(sum / wsum);
}

cv::Mat bf_parallel_naive(const cv::Mat &source, 
    const int diameter, const double sigma_i, const double sigma_s)
{
    cv::Mat dst = cv::Mat::zeros(source.rows, source.cols, CV_8U);
    int radius = diameter / 2;

    // Create an image with a border.
    cv::Mat temp;
    cv::copyMakeBorder(source, temp, radius, radius, radius, radius,
        cv::BorderTypes::BORDER_REFLECT_101);

    // Init color weight.
    double coeff_i = -0.5 / (sigma_i * sigma_i);
    std::vector<double> gi_vec(256);
    double *gi = &gi_vec[0];
    for (int i = 0; i < 256; i++)
        gi[i] = exp(i * i * coeff_i);

    // Generate gaussian space.
    std::vector<double> gs_vec(diameter * diameter);
    std::vector<int> space_coord_vec(diameter * diameter); //< Save here coord.
    double *gs = &gs_vec[0];
    int    *space_coord = &space_coord_vec[0];
    const double coeff_s = -0.5 / (sigma_s * sigma_s);
    int maxk = 0;
    for (int i = -radius; i <= radius; i++)
    {
        for (int j = -radius; j <= radius; j++)
        {
            double r = sqrt(i * i + j * j);
            if (r > radius) //< Circle.
                continue;
            gs[maxk] = exp(r * r * coeff_s);
            space_coord[maxk++] = i * (int)temp.step + j;
        }
    }

    // Copy data to device
    uchar* temp_d;
    handleError(hipMalloc(&temp_d, temp.total()), __LINE__);
    handleError(hipMemcpy(temp_d, temp.data, temp.total(), hipMemcpyHostToDevice), __LINE__);
    uchar* dst_d;
    handleError(hipMalloc(&dst_d, dst.total()), __LINE__);
    handleError(hipMemcpy(dst_d, dst.data, dst.total(), hipMemcpyHostToDevice), __LINE__);
    double* gs_d;
    handleError(hipMalloc(&gs_d, diameter * diameter * sizeof(double)), __LINE__);
    handleError(hipMemcpy(gs_d, gs, diameter * diameter * sizeof(double), hipMemcpyHostToDevice), __LINE__);
    double* gi_d;
    handleError(hipMalloc(&gi_d, 256 * sizeof(double)), __LINE__);
    handleError(hipMemcpy(gi_d, gi, 256 * sizeof(double), hipMemcpyHostToDevice), __LINE__);
    int* space_coord_d;
    handleError(hipMalloc(&space_coord_d, diameter * diameter * sizeof(int)), __LINE__);
    handleError(hipMemcpy(space_coord_d, space_coord, diameter * diameter * sizeof(int), hipMemcpyHostToDevice), __LINE__);
    //Calculate optimal CUDA configuration
    int blockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                        bf_parallel_naive_k, 0, 0);
    // Round up according to matrix size 
    blockSize = (int)sqrt(blockSize);
    dim3 gridSize_2D((source.cols + blockSize - 1) / blockSize, (source.rows + blockSize - 1) / blockSize);
    dim3 blockSize_2D(blockSize, blockSize);
    // Filtering process
    bf_parallel_naive_k << < gridSize_2D, blockSize_2D >> > (temp_d, dst_d, radius, gi_d, gs_d,
                                                space_coord_d, maxk, source.cols,
                                                source.rows, temp.step, dst.step);

    handleError(hipDeviceSynchronize(), __LINE__);
    // Copy data from device
    handleError(hipMemcpy(dst.data, dst_d, dst.total(), hipMemcpyDeviceToHost), __LINE__);
    handleError(hipFree(temp_d), __LINE__);
    handleError(hipFree(dst_d), __LINE__);
    handleError(hipFree(gs_d), __LINE__);
    handleError(hipFree(gi_d), __LINE__);
    handleError(hipFree(space_coord_d), __LINE__);

    return dst;
}